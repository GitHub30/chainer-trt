/*
 * Copyright (c) 2018 Preferred Networks, Inc. All rights reserved.
 */


#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

namespace chainer_trt {
namespace plugin {
    template <typename T>
    __global__ void eltw_sum_kernel(const T* src_gpu, int n_in,
                                    const T* vals_gpu, int n_values,
                                    T* dst_gpu) {
        const int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if(n_in <= idx)
            return;
        const int batch = blockIdx.y;
        const int idx_in_vals = (n_values == 1 ? 0 : idx);
        dst_gpu[batch * n_in + idx] =
          vals_gpu[idx_in_vals] + src_gpu[batch * n_in + idx];
    }

    template <typename T>
    void apply_eltw_sum(const T* src_gpu, int n_in, const T* vals_gpu,
                        int n_values, T* dst_gpu, int batch_size,
                        hipStream_t stream) {
        const int block_size = 1024;
        const int grid_size = (int)std::ceil(1.0 * n_in / block_size);
        dim3 grid(grid_size, batch_size);
        eltw_sum_kernel<T><<<grid, block_size, 0, stream>>>(
          src_gpu, n_in, vals_gpu, n_values, dst_gpu);
    }

    template <typename T>
    __global__ void eltw_sub_kernel(const T* src_gpu, int n_in,
                                    const T* vals_gpu, int n_values,
                                    T* dst_gpu) {
        const int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if(n_in <= idx)
            return;
        const int batch = blockIdx.y;
        const int idx_in_vals = (n_values == 1 ? 0 : idx);
        dst_gpu[batch * n_in + idx] =
          vals_gpu[idx_in_vals] - src_gpu[batch * n_in + idx];
    }

    template <typename T>
    void apply_eltw_sub(const T* src_gpu, int n_in, const T* vals_gpu,
                        int n_values, T* dst_gpu, int batch_size,
                        hipStream_t stream) {
        const int block_size = 1024;
        const int grid_size = (int)std::ceil(1.0 * n_in / block_size);
        dim3 grid(grid_size, batch_size);
        eltw_sub_kernel<T><<<grid, block_size, 0, stream>>>(
          src_gpu, n_in, vals_gpu, n_values, dst_gpu);
    }

    template <typename T>
    __global__ void eltw_mul_kernel(const T* src_gpu, int n_in,
                                    const T* vals_gpu, int n_values,
                                    T* dst_gpu) {
        const int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if(n_in <= idx)
            return;
        const int batch = blockIdx.y;
        const int idx_in_vals = (n_values == 1 ? 0 : idx);
        dst_gpu[batch * n_in + idx] =
          vals_gpu[idx_in_vals] * src_gpu[batch * n_in + idx];
    }

    template <typename T>
    void apply_eltw_mul(const T* src_gpu, int n_in, const T* vals_gpu,
                        int n_values, T* dst_gpu, int batch_size,
                        hipStream_t stream) {
        const int block_size = 1024;
        const int grid_size = (int)std::ceil(1.0 * n_in / block_size);
        dim3 grid(grid_size, batch_size);
        eltw_mul_kernel<T><<<grid, block_size, 0, stream>>>(
          src_gpu, n_in, vals_gpu, n_values, dst_gpu);
    }

    template <typename T>
    __global__ void eltw_div_kernel(const T* src_gpu, int n_in,
                                    const T* vals_gpu, int n_values,
                                    T* dst_gpu) {
        const int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if(n_in <= idx)
            return;
        const int batch = blockIdx.y;
        const int idx_in_vals = (n_values == 1 ? 0 : idx);
        dst_gpu[batch * n_in + idx] =
          vals_gpu[idx_in_vals] / src_gpu[batch * n_in + idx];
    }

    template <typename T>
    void apply_eltw_div(const T* src_gpu, int n_in, const T* vals_gpu,
                        int n_values, T* dst_gpu, int batch_size,
                        hipStream_t stream) {
        const int block_size = 1024;
        const int grid_size = (int)std::ceil(1.0 * n_in / block_size);
        dim3 grid(grid_size, batch_size);
        eltw_div_kernel<T><<<grid, block_size, 0, stream>>>(
          src_gpu, n_in, vals_gpu, n_values, dst_gpu);
    }

    // explicit instantiation (without this, link error will happen)
    template void apply_eltw_sum(const float*, int, const float*, int, float*,
                                 int, hipStream_t);
    template void apply_eltw_sub(const float*, int, const float*, int, float*,
                                 int, hipStream_t);
    template void apply_eltw_mul(const float*, int, const float*, int, float*,
                                 int, hipStream_t);
    template void apply_eltw_div(const float*, int, const float*, int, float*,
                                 int, hipStream_t);
    template void apply_eltw_sum(const __half*, int, const __half*, int,
                                 __half*, int, hipStream_t);
    template void apply_eltw_sub(const __half*, int, const __half*, int,
                                 __half*, int, hipStream_t);
    template void apply_eltw_mul(const __half*, int, const __half*, int,
                                 __half*, int, hipStream_t);
    template void apply_eltw_div(const __half*, int, const __half*, int,
                                 __half*, int, hipStream_t);
}
}
